#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Kernel: Convert RGB image to grayscale
__global__ void rgb2gray(const float *input, float *output, int width, int height) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int totalPixels = width * height;
  if (idx < totalPixels) {
    int rgbIdx = idx * 3;
    float r = input[rgbIdx];
    float g = input[rgbIdx + 1];
    float b = input[rgbIdx + 2];
    output[idx] = 0.21f * r + 0.71f * g + 0.07f * b;
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */
  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);
  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3 (RGB channels)
  imageChannels = wbImage_getChannels(inputImage);

  // Since the output image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  // Allocate GPU memory
  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  // Copy host memory to GPU
  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  // Launch the kernel to convert the image to grayscale
  wbTime_start(Compute, "Doing the computation on the GPU");
  int numPixels = imageWidth * imageHeight;
  int blockSize = 256;
  int gridSize = (numPixels + blockSize - 1) / blockSize;
  rgb2gray<<<gridSize, blockSize>>>(deviceInputImageData, deviceOutputImageData,
                                    imageWidth, imageHeight);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  // Copy the result from GPU to CPU
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  // Free allocated GPU memory
  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}