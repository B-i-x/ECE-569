#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#ifndef TILE_WIDTH
#define TILE_WIDTH 16
#endif

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  // Define shared memory tiles for A and B.
  __shared__ float As[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

  // Identify the block and thread indices.
  int bx = blockIdx.x; 
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Calculate the row and column index of the C element to work on.
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  // Accumulate the dot product in a register.
  float sum = 0.0f;

  // Loop over all the tiles needed to cover the A matrix's columns / B matrix's rows.
  int numTiles = (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH;
  for (int t = 0; t < numTiles; t++) {
    // Load the tile from A into shared memory.
    int tiledACol = t * TILE_WIDTH + tx;
    if (row < numARows && tiledACol < numAColumns)
      As[ty][tx] = A[row * numAColumns + tiledACol];
    else
      As[ty][tx] = 0.0f;  // Pad with zeros if out of bounds

    // Load the tile from B into shared memory.
    int tiledBRow = t * TILE_WIDTH + ty;
    if (tiledBRow < numBRows && col < numBColumns)
      Bs[ty][tx] = B[tiledBRow * numBColumns + col];
    else
      Bs[ty][tx] = 0.0f;  // Pad with zeros if out of bounds

    // Synchronize to ensure all threads have loaded their data.
    __syncthreads();

    // Multiply the two tiles together and accumulate the results.
    for (int k = 0; k < TILE_WIDTH; k++) {
      sum += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to ensure that computation is done before loading new tiles.
    __syncthreads();
  }

  // Write the computed value to C if within matrix bounds.
  if (row < numCRows && col < numCColumns) {
    C[row * numCColumns + col] = sum;
  }
}


int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA; // A matrix on device
  float *deviceB; // B matrix on device
  float *deviceC; // C matrix on device
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  hipEvent_t astartEvent, astopEvent;
  float aelapsedTime;
  hipEventCreate(&astartEvent);
  hipEventCreate(&astopEvent);

  args = wbArg_read(argc, argv);

 //ali  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;  // set to correct value
  numCColumns = numBColumns;  // set to correct value
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

 //ali  wbTime_stop(Generic, "Importing data and creating memory on host");

//ali  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
//ali  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
//ali  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  
 //ali  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here for A, B and C
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));
  
  
 //ali wbTime_stop(GPU, "Allocating GPU memory.");

 //ali wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here for A and B
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float),hipMemcpyHostToDevice);
  
  
 //ali wbTime_stop(GPU, "Copying input memory to the GPU.");

  
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDim((numCColumns + TILE_WIDTH - 1) / TILE_WIDTH,
               (numCRows + TILE_WIDTH - 1) / TILE_WIDTH,
               1);
    hipEventRecord(astartEvent, 0);

    matrixMultiplyShared<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC,
    numARows, numAColumns,
    numBRows, numBColumns,
    numCRows, numCColumns);
    hipDeviceSynchronize();

    hipEventRecord(astopEvent, 0);
    hipEventSynchronize(astopEvent);
    hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
    printf("Total execution time (ms) %f for block size %d x %d matrix size of %d x %d and %d x %d\n",aelapsedTime,TILE_WIDTH,TILE_WIDTH,numARows,numAColumns,numBRows,numBColumns);
    // ali: this loop will sweep blocks 4x4, 8x8, 16x16 and 32x32
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float),hipMemcpyDeviceToHost);


    wbSolution(args, hostC, numCRows, numCColumns);

    //ali  wbTime_start(GPU, "Freeing GPU Memory");

    //@@ Free the GPU memory here
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    //ali  wbTime_stop(GPU, "Freeing GPU Memory");
    //ali  wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
