#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  // Calculate the row index of the C element to work on
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  // Calculate the column index of the C element to work on
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Make sure the thread is within bounds of C
  if (row < numCRows && col < numCColumns) {
    float sum = 0.0f;
    // Loop over the A row and B column to compute the dot product
    for (int k = 0; k < numAColumns; ++k) {
      sum += A[row * numAColumns + k] * B[k * numBColumns + col];
    }
    // Write the computed value to C
    C[row * numCColumns + col] = sum;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA; // A matrix on device
  float *deviceB; // B matrix on device
  float *deviceC; // C matrix on device
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;   // set to correct value
  numCColumns = numBColumns;   // set to correct value

  // Allocate memory for the output matrix C on the host
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  
  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here for A, B and C
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here for A and B
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float),hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  // set block size to 16,16 and determine the grid dimensions
  // use dim3 structure for setting block and grid dimensions
  // Set block size to 16x16 and compute grid dimensions so that each thread computes one element in C
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((numCColumns + blockDim.x - 1) / blockDim.x,
               (numCRows + blockDim.y - 1) / blockDim.y,
               1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC,
    numARows, numAColumns,
    numBRows, numBColumns,
    numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float),hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
